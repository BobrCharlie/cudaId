#include "hip/hip_runtime.h"
#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#else
#define KERNEL_ARGS2(grid, block)
#endif
#define pic 3.14159265359
#include<ctime>
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <windows.h>

using namespace cv;


void setDev()
{
	hipSetDevice(0);
}

void setDev(int i)
{
	hipSetDevice(i);
}

void delDev()
{
	hipDeviceReset();
}

__global__ void GradKernel(char *image, char *cont, unsigned int sizex, int sizey, int rg) //360/8 x2+y2=r2 r and x
{
	int x = threadIdx.x;
	int y = blockIdx.x;
	int maxim = 0; int maxint;
	/*if (y != 0 && y != sizey && x != 0 && x != sizex)
	{

		if (image[(y - 1)*sizex + x - 1] - image[y*sizex + x] > maxim)
		{
			maxim = image[(y - 1)*sizex + x - 1] - image[y*sizex + x];
			maxint = pic+pic/2+pic/4;
		}

		if (image[(y - 1)*sizex + x] - image[y*sizex + x] > maxim)
		{
			maxim = image[(y - 1)*sizex + x] - image[y*sizex + x];
			maxint = 0;
		}

		if (image[(y - 1)*sizex + x + 1] - image[y*sizex + x] > maxim)
		{
			maxim = image[(y - 1)*sizex + x + 1] - image[y*sizex + x];
			maxint = pic/4;
		}

		if (image[y*sizex + x - 1] - image[y*sizex + x] > maxim)
		{
			maxim = image[y*sizex + x - 1] - image[y*sizex + x];
			maxint = pic/2;
		}

		if (image[y*sizex + x + 1] - image[y*sizex + x] > maxim)
		{
			maxim = image[y*sizex + x + 1] - image[y*sizex + x];
			maxint = pic/2+pic/4;
		}

		if (image[(y + 1)*sizex + x - 1] - image[y*sizex + x] > maxim)
		{
			maxim = image[(y + 1)*sizex + x - 1] - image[y*sizex + x];
			maxint = pic;
		}
		if (image[(y + 1)*sizex + x] - image[y*sizex + x] > maxim)
		{
			maxim = image[(y + 1)*sizex + x] - image[y*sizex + x];
			maxint = pic+pic/4;
		}

		if (image[(y + 1)*sizex + x + 1] - image[y*sizex + x] > maxim)
		{
			maxim = image[(y + 1)*sizex + x + 1] - image[y*sizex + x];
			maxint = pic+pic/2;
		}
	}*/
	//cont[3 * (y*sizex + x)] = maxim;
	//cont[3 * (y*sizex + x) + 1] = sinf(maxint) * 255;
	//cont[3 * (y*sizex + x) + 2] = image[y*sizex + x];
	if (y > rg && y < sizey-rg && x > rg && x < sizex-rg)
	{
		float xk = x - rg;
		float y1 = sqrt((float)(xk*xk + rg*rg));
		float y2 = -y1;
		while (xk < x)
		{
			float xf = x - 0.5;
			int xi = int(xf);
			int yfromx;
			yfromx = int((xf - xk) * (y1 - y) / (xk - x) + y);
			while ((image[yfromx*sizex + xi] > image[y*sizex + x]) && (xf > (x - rg)))
			{
				xf = xf - 0.5;
				xi = int(xf);
				yfromx = int((xf - xk)*(y1 - y) / (xk - x) + y);
			}
			if (xf == x - rg)
				if (image[yfromx*sizex + xi] > cont[3 * y*sizex + x])
				{
					cont[3 * y*sizex + x] = image[yfromx*sizex + xi] - image[y*sizex + x];
				}

			xk += 0.5;
			float y1 = sqrtf((float)(xk*xk + rg*rg));
			float y2 = -y1;
		}
	}
	/*for (int t = y - rg; t < y + rg + 1; t++)
	{

	float xf = x - 0.5;
	int xi = int(xf)-1;
	int yfromx;
	yfromx = int((xf - x + rg)*(y - t) / rg + t);
	while ((image[yfromx*sizex + xi] > image[y*sizex + x]) && (xf > (x - rg))) //
	{
	xf = xf - 0.5;
	xi = int(xf)-1;
	if (t>y)
	yfromx = int((xf - x + rg)*(y - t) / rg + t);
	else if (t<y)
	yfromx = int((xf - x + rg)*(y - t) / rg + t) + 1;
	else yfromx = y;
	}
	if (xf == x - rg)
	if (image[yfromx*sizex + xi] > cont[y*sizex + x])
	{
	cont[y*sizex + x] = image[yfromx*sizex + xi] - image[y*sizex + x];
	}

	xf = x + 0.5;
	xi = int(xf) + 1;
	yfromx = int((xf - x - rg)*(y - t) / (-rg) + t) + 1;
	while ((image[yfromx*sizex + xi] > image[y*sizex + x]) && (xf < x + rg))
	{
	xf = xf + 0.5;
	xi = int(xf) + 1;
	if (t>y)
	yfromx = int((xf - x + rg)*(y - t) / rg + t);
	else if (t<y)
	yfromx = int((xf - x + rg)*(y - t) / rg + t) + 1;
	else yfromx = y;
	}
	if (xf == x + rg)
	if (image[yfromx*sizex + xi] > cont[y*sizex + x])
	{
	cont[y*sizex + x] = image[yfromx*sizex + xi] - image[y*sizex + x];
	}

	/*xf = x - 0.5;
	xi = int(xf);
	yfromx = (xf - t)*(-rg) / (x - t) + y - rg;
	while ((image[yfromx*sizex + xi] > image[y*sizex + x]) && (xf > x - rg))
	{
	xf = xf - 0.5;
	xi = int(xf);
	}
	if (xf == x - rg)
	if ((image[yfromx*sizex + xi] > cont[y*sizex + x]))
	{
	cont[y*sizex + x] = image[yfromx*sizex + xi] - image[y*sizex + x];
	}

	xf = x + 0.5;
	xi = int(xf) + 1;
	yfromx = (xf - t)*rg / (x - t) + y + rg;
	while ((image[yfromx*sizex + xi] > image[y*sizex + x]) && (xf < x + rg))
	{
	xf = xf + 0.5;
	xi = int(xf) + 1;
	}
	if (xf == x + rg)
	if (image[yfromx*sizex + xi] > cont[y*sizex + x])
	{
	cont[y*sizex + x] = image[yfromx*sizex + xi] - image[y*sizex + x];
	}
	}*/
}

void findGrad(char *image, char *cont, unsigned int sizex, unsigned int sizey, int rg)
{
	char *gray_d;
	char *cont_d;
	hipMalloc((void**)&gray_d, sizex * sizey * sizeof(char));
	hipMalloc((void**)&cont_d, sizex * sizey * 3 * sizeof(char));

	hipMemcpy(gray_d, image, sizex * sizey * sizeof(char), hipMemcpyHostToDevice);

	GradKernel KERNEL_ARGS2(dim3(sizey), dim3(sizex)) (gray_d, cont_d, sizex, sizey, rg);
	hipDeviceSynchronize();
	hipMemcpy(cont, cont_d, sizex * sizey * 3 * sizeof(char), hipMemcpyDeviceToHost);

	hipFree(cont_d);
	hipFree(gray_d);
}
int main()
{
	VideoCapture capture(0);
	Mat gray, image, cont;
	int width = 640;
	int height = 480;
	int rg;
	std::cout << "gradient radius: "; std::cin >> rg;
	setDev(0);
	cont = Mat::Mat(Size(width, height), CV_8UC3);
	time_t t1;
	while (1)
	{
		capture >> image;
		//imshow("image", image);		//CV_8UC3; Vec3b;
		cvtColor(image, gray, CV_BGR2GRAY);
		//imshow("gray", gray); 		//Canny(gray, cont, 50, 100);
		t1 = clock();
		findGrad((char*)gray.data, (char*)cont.data, width, height, rg);
		std::cout << clock() - t1 << std::endl;
		imshow("cont1", cont);
		if (waitKey(33) == 27)
		{
			delDev();
			imwrite("constMy.png", cont);
			return 1;
		}

		//	imshow("cont2", cont);
	}
	delDev();
	return 1;
}

/*
ParallelDevice::ParallelDevice()
{
hipSetDevice(0);
}

ParallelDevice::ParallelDevice(int i)
{
hipSetDevice(i);
}

ParallelDevice::~ParallelDevice()
{
hipDeviceReset();
}*/

/*	cudaStatus = hipDeviceSynchronize();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
goto Error;
}*/

/*	cudaStatus = hipMemcpy(c, dev_c, size * size * sizeof(int), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipMemcpy failed!");
goto Error;
}*/

/*
Error:
hipFree(dev_c);
hipFree(dev_a);
hipFree(dev_b);

return cudaStatus; */

/*int main()
{
const int size = 100; int i, j;
int a[size][size];
int b[size][size];
int c[size][size];

for (i = 0; i < size; i++)
for (j = 0; j < size; j++)
{
a[i][j] = rand()%1000;
b[i][j] = rand()%1000;
}


for (i = 0; i < size; i++)
for (j = 0; j < size; j++)
{
c[i][j] = 1;
}

// Add vectors in parallel.
hipError_t cudaStatus = addWithCuda(*c, *a, *b, size);
if (cudaStatus != hipSuccess) {
fprintf(stderr, "addWithCuda failed!");
return 1;
}

// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
cudaStatus = hipDeviceReset();
if (cudaStatus != hipSuccess) {
fprintf(stderr, "hipDeviceReset failed!");
return 1;
}

return 0;
}*/